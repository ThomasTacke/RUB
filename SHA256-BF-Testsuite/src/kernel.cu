#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */

#include "sha256gpu.h"
#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdlib.h>
#include <stdio.h>
#include <string>
#include <iostream>
#include <iomanip>
#include <math.h>
#include ""
#include "hip/hip_runtime.h"

#include "sha256cpu.h"

using namespace std;
using std::fixed;

#define maxLength 8
#define minLength 1

void clearScreen() {
#ifdef _WIN32
	std::system ( "CLS" );
#else
	// Assume POSIX
	std::system("clear");
#endif
}

__constant__ int K[64] = { 0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5,
		0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5, 0xd807aa98, 0x12835b01,
		0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
		0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa,
		0x5cb0a9dc, 0x76f988da, 0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7,
		0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967, 0x27b70a85, 0x2e1b2138,
		0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
		0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624,
		0xf40e3585, 0x106aa070, 0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5,
		0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3, 0x748f82ee, 0x78a5636f,
		0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2 };
__constant__ uint32_t h_values[] = { 0x6a09e667, 0xbb67ae85, 0x3c6ef372,
		0xa54ff53a, 0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19 };

__constant__ char usedAlphabet[] = { 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h',
		'i', 'j', 'k', 'l', 'm', 'n', 'o', 'p', 'q', 'r', 's', 't', 'u', 'v',
		'w', 'x', 'y', 'z' };
__constant__ uint16_t usedAlphabetSize = sizeof(usedAlphabet);

__global__ void kernel(uint32_t *target, char *dev_result,
		char *devPositionInUsedAlphabet, uint16_t iterations_per_thread,
		char *dev_match_found, uint8_t stringLength, uint8_t usedAlphabetSize) {
	uint8_t stringLengthTemp = stringLength;
	uint32_t S0;
	uint32_t S1;
	unsigned char block[64];
	uint32_t W[64];
	bool match = false;
	int thread_number = threadIdx.x + blockIdx.x * blockDim.x;

	/*copy starting positionInUsedAlphabet*/
	char attackedString[maxLength + 1];
	char positionInUsedAlphabet[maxLength];
	for (int i = 0; i < stringLength + 1; i++) {
		attackedString[i] = 0;
	}
	for (int i = 0; i < stringLength; i++) {
		positionInUsedAlphabet[i] = devPositionInUsedAlphabet[i];
	}

	/*set the position for each thread individually*/
	//working wieder EINBAUEN falls neuer Ansatz nicht geht
	/*for(int j=stringLength-1;j>=0;j--)
	 {
	 uint64_t temp=1;
	 //manual pow for pow(usedAlphabetSize, stringLength-j)
	 for(int i=1;i<stringLength-j;i++)
	 {
	 temp=temp*usedAlphabetSize;
	 }
	 positionInUsedAlphabet[j]=positionInUsedAlphabet[j]+((uint32_t)(iterations_per_thread*thread_number/ temp) %usedAlphabetSize);
	 //positionInUsedAlphabet[j]=positionInUsedAlphabet[j]+((uint32_t)(iterations_per_thread*thread_number/ pow( (double) usedAlphabetSize, (double) (stringLength-j) )) %usedAlphabetSize);
	 }

	 /*set the position for each thread individually*/

	float temp = iterations_per_thread * thread_number;
	for (int j = stringLength - 1; j >= 0; j--) {
		positionInUsedAlphabet[j] = positionInUsedAlphabet[j]
				+ (uint32_t(temp) % usedAlphabetSize);
		temp = temp / usedAlphabetSize;
		if (temp < 1)
			break;
	}
	positionInUsedAlphabet[0] = positionInUsedAlphabet[0]
			+ (uint32_t(temp) % usedAlphabetSize);
	if (temp > usedAlphabetSize) {
		temp = iterations_per_thread * thread_number;
		return;
	}
	for (int j = stringLength - 1; j > 0; j--) {
		while (positionInUsedAlphabet[j] >= usedAlphabetSize) {
			positionInUsedAlphabet[j] -= usedAlphabetSize;
			positionInUsedAlphabet[j - 1]++;
		}
	}

	//temporary variables, values irrelevant*/
	uint32_t hashes[8];

	for (int j = 0; j < iterations_per_thread; j++) {

		for (int i = stringLength - 1; i > 0; i--) {
			if (positionInUsedAlphabet[i] >= usedAlphabetSize) {
				positionInUsedAlphabet[i] = 0;
				positionInUsedAlphabet[i - 1]++;
			}
		}
		if (positionInUsedAlphabet[0] >= usedAlphabetSize)
			break; //if(positionInUsedAlphabet[maxLength-stringLength]>usedAlphabetSize-1)break;
		for (int i = 0; i < stringLength; i++) {
			attackedString[i] = usedAlphabet[positionInUsedAlphabet[i]];
		}

		//Message Copy
		for (int i = 0; i < stringLength; i++) {
			block[i] = attackedString[i];
		}

		//SHA2-begin
		//Padding
		block[stringLength] = 0x80;	//append a 1 (1000.0000)

		for (int i = stringLength + 1; i < 56; i++)	//fill with 0 except the last 8-byte
				{
			block[i] = 0x0;
		}

		stringLength = stringLength * 8;
		//append the length of the attackedString in big endianes
		for (int i = 0; i < 8; i++) {
			block[63 - i] = stringLength;
			stringLength = stringLength >> 8;
		}

		uint32_t a = h_values[0];
		uint32_t b = h_values[1];
		uint32_t c = h_values[2];
		uint32_t d = h_values[3];
		uint32_t e = h_values[4];
		uint32_t f = h_values[5];
		uint32_t g = h_values[6];
		uint32_t h = h_values[7];
		uint32_t T1;
		uint32_t T2;

		//compute W
		for (int i = 0; i < 16; i++) {
			W[i] = (block[i * 4] << 24) | (block[i * 4 + 1] << 16)
					| (block[i * 4 + 2] << 8) | (block[i * 4 + 3]);
		}
		for (int i = 16; i < 64; i++) {
			S0 =
					(((W[i - 15] >> 7) | (W[i - 15] << 25))
							^ ((W[i - 15] >> 18) | (W[i - 15] << 14))
							^ (W[i - 15] >> 3));
			S1 = (((W[i - 2] >> 17) | (W[i - 2] << 15))
					^ ((W[i - 2] >> 19) | (W[i - 2] << 13)) ^ (W[i - 2] >> 10));
			W[i] = (S0 + S1 + W[i - 7] + W[i - 16]) & 0xFFFFFFFF;
		}

		//Hash
		for (int i = 0; i < 64; i++) {
			T1 = (h + ((e & f) ^ ((~e) & g) /*Ch(e,f,g)*/)
					+ (((e >> 6) | (e << 26)) ^ ((e >> 11) | (e << 21))
							^ ((e >> 25) | (e << 7)) /*Sigma1(e)*/) + K[i]
					+ W[i]) & 0xFFFFFFFF;
			T2 = ((((a >> 2) | (a << 30)) ^ ((a >> 13) | (a << 19))
					^ ((a >> 22) | (a << 10)) /*Sigma0(a)*/)
					+ ((a & b) ^ (a & c) ^ (b & c) /*Maj(a,b,c)*/))
					& 0xFFFFFFFF;
			h = g;
			g = f;
			f = e;
			e = (d + T1) & 0xFFFFFFFF;
			d = c;
			c = b;
			b = a;
			a = (T1 + T2) & 0xFFFFFFFF;
		}

		hashes[0] = (a + h_values[0]) & 0xFFFFFFFF;
		hashes[1] = (b + h_values[1]) & 0xFFFFFFFF;
		hashes[2] = (c + h_values[2]) & 0xFFFFFFFF;
		hashes[3] = (d + h_values[3]) & 0xFFFFFFFF;
		hashes[4] = (e + h_values[4]) & 0xFFFFFFFF;
		hashes[5] = (f + h_values[5]) & 0xFFFFFFFF;
		hashes[6] = (g + h_values[6]) & 0xFFFFFFFF;
		hashes[7] = (h + h_values[7]) & 0xFFFFFFFF;

		//SHA2-end

		match = true;
		for (int i = 0; i < 8; i++) {
			if (hashes[i] != target[i]) {
				match = false;
			}
		}
		if (match) {
			for (int i = 0; i < sizeof(dev_result); i++) {
				dev_result[i] = 0;
			}
			for (int i = 0; i < stringLengthTemp; i++) {
				dev_result[i] = attackedString[i];
			}
			dev_match_found[0] = 1;
			return;
		}
		stringLength = stringLengthTemp;
		positionInUsedAlphabet[stringLength - 1]++;
	}

}

/*
 Start the kernel with 100 iterations and measure the time. Then adjust the iterations_per_thread in a way that the kernel will take 2 seconds
 */
int get_iterations_per_thread(uint16_t number_of_blocks,
		uint16_t threads_per_block) {
	uint32_t impossible_target[] = { 0x183bddb1, 0xf21ab681, 0x3bd9b6b7,
			0x907bfe76, 0x49036ad7, 0xbc75cd39, 0x6df352fe, 0xd7dcb135 };
	uint32_t *dev_impossible_target;
	char *dev_result;
	char *dev_match_found;
	char *devPositionInUsedAlphabet;
	char match_found[1];
	float elapsedTime_kernel;

	char usedAlphabet[] = { 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h', 'i', 'j',
			'k', 'l', 'm', 'n', 'o', 'p', 'q', 'r', 's', 't', 'u', 'v', 'w',
			'x', 'y', 'z' };
	uint8_t usedAlphabetSize = sizeof(usedAlphabet);
	char *dev_usedAlphabet;

	hipEvent_t start_kernel, stop_kernel;
	HANDLE_ERROR(hipEventCreate(&start_kernel));
	HANDLE_ERROR(hipEventCreate(&stop_kernel));

	HANDLE_ERROR(hipMalloc((void**)&dev_result,maxLength));
	HANDLE_ERROR(hipMalloc((void** )&dev_match_found, 1));
	HANDLE_ERROR(
			hipMemcpy(dev_match_found, match_found, 1,
					hipMemcpyHostToDevice));

	HANDLE_ERROR(
			hipMalloc((void** )&dev_impossible_target, sizeof(uint32_t) * 8));
	HANDLE_ERROR(
			hipMemcpy(dev_impossible_target, impossible_target,
					sizeof(uint32_t) * 8, hipMemcpyHostToDevice));

	HANDLE_ERROR(hipMalloc((void** )&dev_usedAlphabet, usedAlphabetSize));
	HANDLE_ERROR(
			hipMemcpy(dev_usedAlphabet, usedAlphabet, usedAlphabetSize,
					hipMemcpyHostToDevice));

	char * startString = (char*) malloc(sizeof(char) * (11));

	HANDLE_ERROR(hipEventRecord(start_kernel, 0));

	kernel<<<number_of_blocks, threads_per_block>>>(dev_impossible_target,
			dev_result, devPositionInUsedAlphabet, 2000, dev_match_found, 10,
			usedAlphabetSize);

	HANDLE_ERROR(hipEventRecord(stop_kernel, 0));
	HANDLE_ERROR(hipEventSynchronize(stop_kernel));
	HANDLE_ERROR(
			hipEventElapsedTime(&elapsedTime_kernel, start_kernel,
					stop_kernel));

	cout << endl << elapsedTime_kernel << "=> "
			<< 110 * 1000 * 2 / elapsedTime_kernel / 4;	//factor 4 is because of 4 streams  90 klappt f�r release, f�r profiling wird auf 70 reduziert werden

	/*string schwts = "";
	 cin>>schwts;*/

	hipFree(dev_usedAlphabet);
	hipFree(dev_result);
	hipFree(dev_match_found);
	hipFree(dev_impossible_target);

	return (int) (110 * 1000 * 2 / elapsedTime_kernel / 4);	//factor 4 is because of 4 streams  90 klappt f�r release, f�r profiling wird auf 70 reduziert werden
}

void sha256bruteforce(uint32_t *target) {
	//Get cuda device prop
	hipDeviceProp_t prop;
	HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));
	uint32_t max_threads_per_SM = prop.maxThreadsPerMultiProcessor;
	uint32_t max_blocks_per_SM = 16;
	if (prop.major < 3)
		max_blocks_per_SM = 8;
	uint32_t threads_per_block = max_threads_per_SM / max_blocks_per_SM;
	uint32_t number_of_blocks = max_blocks_per_SM * prop.multiProcessorCount;
	uint32_t *dev_target;
	char result[maxLength + 1];
	char *dev_result;
	char *dev_match_found;
	char match_found[1];

	char usedAlphabet[] = { 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h', 'i', 'j',
			'k', 'l', 'm', 'n', 'o', 'p', 'q', 'r', 's', 't', 'u', 'v', 'w',
			'x', 'y', 'z' };
	uint8_t usedAlphabetSize = sizeof(usedAlphabet);
	char *dev_usedAlphabet;
	char *devPositionInUsedAlphabet;
	match_found[0] = 0;
	hipStream_t stream0, stream1, stream2, stream3;

	// capture the start time
	hipEvent_t start_kernel, start_overall, stop_kernel, stop_overall;
	HANDLE_ERROR(hipEventCreate(&start_kernel));
	HANDLE_ERROR(hipEventCreate(&stop_kernel));
	HANDLE_ERROR(hipEventCreate(&start_overall));
	HANDLE_ERROR(hipEventCreate(&stop_overall));

	HANDLE_ERROR(hipMalloc((void** )&dev_result, 16));
	HANDLE_ERROR(hipMalloc((void** )&dev_match_found, 1));
	HANDLE_ERROR(
			hipMemcpy(dev_match_found, match_found, 1,
					hipMemcpyHostToDevice));

	HANDLE_ERROR(hipMalloc((void** )&dev_target, sizeof(uint32_t) * 8));
	HANDLE_ERROR(
			hipMemcpy(dev_target, target, sizeof(uint32_t) * 8,
					hipMemcpyHostToDevice));

	HANDLE_ERROR(hipMalloc((void** )&dev_usedAlphabet, usedAlphabetSize));
	HANDLE_ERROR(
			hipMemcpy(dev_usedAlphabet, usedAlphabet, usedAlphabetSize,
					hipMemcpyHostToDevice));

	uint32_t iterations_per_thread = 9000;
	//uint32_t iterations_per_thread=get_iterations_per_thread(number_of_blocks,threads_per_block);	

	float elapsedTime_overall;

	HANDLE_ERROR(hipStreamCreate(&stream0));
	HANDLE_ERROR(hipStreamCreate(&stream1));
	HANDLE_ERROR(hipStreamCreate(&stream2));
	HANDLE_ERROR(hipStreamCreate(&stream3));
	HANDLE_ERROR(hipEventRecord(start_overall, 0));

	double temp = number_of_blocks * threads_per_block * iterations_per_thread;
	cout << "Iterations per thread: " << iterations_per_thread << endl
			<< "threads per block: " << threads_per_block << endl
			<< "number of blocks: " << number_of_blocks << endl
			<< "=>steps per kernel: "
			<< number_of_blocks * threads_per_block * iterations_per_thread
			<< endl;
	for (int i = minLength; i <= maxLength; i++)//i= length of the attacked String
			{
		cout << endl << endl << "Length " << i << endl;
		char * startString = (char*) malloc(sizeof(char) * (i + 1));//i = length of the string
		uint8_t * positionInUsedAlphabet = (uint8_t*) malloc(sizeof(char) * i);
		//fill the startString with the first character in the used alphabet
		for (int j = 0; j < i; j++) {
			positionInUsedAlphabet[j] = 0;
			startString[j] = usedAlphabet[0];
		}
		startString[i] = 0;	//terminate with 0

		HANDLE_ERROR(
				hipMalloc((void** )&devPositionInUsedAlphabet,
						sizeof(uint8_t) * i));
		//HANDLE_ERROR( hipEventRecord( start_kernel, 0 ) );
		do {

			HANDLE_ERROR(
					hipMemcpy(devPositionInUsedAlphabet,
							positionInUsedAlphabet, sizeof(uint8_t) * i,
							hipMemcpyHostToDevice));

			cout << "attacked String: " << startString << endl;
			//HANDLE_ERROR( hipEventRecord( start_kernel, 0 ) );
			if (match_found[0] == 1) {
				break;
			}
			kernel<<<number_of_blocks, threads_per_block, 0, stream0>>>(
					dev_target, dev_result, devPositionInUsedAlphabet,
					iterations_per_thread, dev_match_found, i,
					usedAlphabetSize);
			/*HANDLE_ERROR( hipEventRecord( stop_kernel, 0 ) );
			 HANDLE_ERROR( hipEventSynchronize( stop_kernel ) );
			 HANDLE_ERROR( hipEventElapsedTime( &elapsedTime_kernel,start_kernel, stop_kernel ) );
			 cout<<elapsedTime_kernel<<endl;*/
			//change the attacked string for the second kernel
			for (int j = i - 1; j > 0; j--) {
				positionInUsedAlphabet[j] = positionInUsedAlphabet[j]
						+ (uint32_t(temp) % usedAlphabetSize);
				temp = temp / usedAlphabetSize;
				if (temp < 1)
					break;
			}
			positionInUsedAlphabet[0] = positionInUsedAlphabet[0]
					+ (uint32_t(temp) % usedAlphabetSize);
			if (temp > usedAlphabetSize) {
				temp = number_of_blocks * threads_per_block
						* iterations_per_thread;
				break;
			}
			for (int j = i - 1; j > 0; j--) {
				while (positionInUsedAlphabet[j] >= usedAlphabetSize) {
					positionInUsedAlphabet[j] -= usedAlphabetSize;
					positionInUsedAlphabet[j - 1]++;
				}
			}
			temp = number_of_blocks * threads_per_block * iterations_per_thread;
			for (int j = 0; j < i; j++) {
				startString[j] = usedAlphabet[positionInUsedAlphabet[j]];
			}
			HANDLE_ERROR(
					hipMemcpy(devPositionInUsedAlphabet,
							positionInUsedAlphabet, sizeof(uint8_t) * i,
							hipMemcpyHostToDevice));
			cout << "attacked String: " << startString << endl;
			if (positionInUsedAlphabet[0] >= usedAlphabetSize)
				break;
			match_found[0] = 0;
			HANDLE_ERROR(
					hipMemcpy(match_found, dev_match_found, 1,
							hipMemcpyDeviceToHost));
			if (match_found[0] == 1) {
				break;
			}
			kernel<<<number_of_blocks, threads_per_block, 0, stream1>>>(
					dev_target, dev_result, devPositionInUsedAlphabet,
					iterations_per_thread, dev_match_found, i,
					usedAlphabetSize);
			//change the attacked string for the third kernel
			for (int j = i - 1; j > 0; j--) {
				positionInUsedAlphabet[j] = positionInUsedAlphabet[j]
						+ (uint32_t(temp) % usedAlphabetSize);
				temp = temp / usedAlphabetSize;
				if (temp < 1)
					break;
			}
			positionInUsedAlphabet[0] = positionInUsedAlphabet[0]
					+ (uint32_t(temp) % usedAlphabetSize);
			if (temp > usedAlphabetSize) {
				temp = number_of_blocks * threads_per_block
						* iterations_per_thread;
				break;
			}
			for (int j = i - 1; j > 0; j--) {
				while (positionInUsedAlphabet[j] >= usedAlphabetSize) {
					positionInUsedAlphabet[j] -= usedAlphabetSize;
					positionInUsedAlphabet[j - 1]++;
				}
			}
			temp = number_of_blocks * threads_per_block * iterations_per_thread;
			for (int j = 0; j < i; j++) {
				startString[j] = usedAlphabet[positionInUsedAlphabet[j]];
			}
			HANDLE_ERROR(
					hipMemcpy(devPositionInUsedAlphabet,
							positionInUsedAlphabet, sizeof(uint8_t) * i,
							hipMemcpyHostToDevice));
			cout << "attacked String: " << startString << endl;
			if (positionInUsedAlphabet[0] >= usedAlphabetSize)
				break;
			match_found[0] = 0;
			HANDLE_ERROR(
					hipMemcpy(match_found, dev_match_found, 1,
							hipMemcpyDeviceToHost));
			if (match_found[0] == 1) {
				break;
			}
			kernel<<<number_of_blocks, threads_per_block, 0, stream2>>>(
					dev_target, dev_result, devPositionInUsedAlphabet,
					iterations_per_thread, dev_match_found, i,
					usedAlphabetSize);
			//change the attacked string for the fourth kernel
			for (int j = i - 1; j > 0; j--) {
				positionInUsedAlphabet[j] = positionInUsedAlphabet[j]
						+ (uint32_t(temp) % usedAlphabetSize);
				temp = temp / usedAlphabetSize;
				if (temp < 1)
					break;
			}
			positionInUsedAlphabet[0] = positionInUsedAlphabet[0]
					+ (uint32_t(temp) % usedAlphabetSize);
			if (temp > usedAlphabetSize) {
				temp = number_of_blocks * threads_per_block
						* iterations_per_thread;
				break;
			}
			for (int j = i - 1; j > 0; j--) {
				while (positionInUsedAlphabet[j] >= usedAlphabetSize) {
					positionInUsedAlphabet[j] -= usedAlphabetSize;
					positionInUsedAlphabet[j - 1]++;
				}
			}
			temp = number_of_blocks * threads_per_block * iterations_per_thread;
			for (int j = 0; j < i; j++) {
				startString[j] = usedAlphabet[positionInUsedAlphabet[j]];
			}
			HANDLE_ERROR(
					hipMemcpy(devPositionInUsedAlphabet,
							positionInUsedAlphabet, sizeof(uint8_t) * i,
							hipMemcpyHostToDevice));
			cout << "attacked String: " << startString << endl;
			if (positionInUsedAlphabet[0] >= usedAlphabetSize)
				break;
			match_found[0] = 0;
			HANDLE_ERROR(
					hipMemcpy(match_found, dev_match_found, 1,
							hipMemcpyDeviceToHost));
			if (match_found[0] == 1) {
				break;
			}
			kernel<<<number_of_blocks, threads_per_block, 0, stream3>>>(
					dev_target, dev_result, devPositionInUsedAlphabet,
					iterations_per_thread, dev_match_found, i,
					usedAlphabetSize);
			//change the attacked string for the fist kernel
			for (int j = i - 1; j > 0; j--) {
				positionInUsedAlphabet[j] = positionInUsedAlphabet[j]
						+ (uint32_t(temp) % usedAlphabetSize);
				temp = temp / usedAlphabetSize;
				if (temp < 1)
					break;
			}
			positionInUsedAlphabet[0] = positionInUsedAlphabet[0]
					+ (uint32_t(temp) % usedAlphabetSize);
			if (temp > usedAlphabetSize) {
				temp = number_of_blocks * threads_per_block
						* iterations_per_thread;
				break;
			}
			for (int j = i - 1; j > 0; j--) {
				while (positionInUsedAlphabet[j] >= usedAlphabetSize) {
					positionInUsedAlphabet[j] -= usedAlphabetSize;
					positionInUsedAlphabet[j - 1]++;
				}
			}
			for (int j = 0; j < i; j++) {
				startString[j] = usedAlphabet[positionInUsedAlphabet[j]];
			}
			temp = number_of_blocks * threads_per_block * iterations_per_thread;
			if (positionInUsedAlphabet[0] >= usedAlphabetSize)
				break;
			match_found[0] = 0;
			HANDLE_ERROR(
					hipMemcpy(match_found, dev_match_found, 1,
							hipMemcpyDeviceToHost));
			if (match_found[0] == 1) {
				break;
			}
		} while (1);
		if (match_found[0] == 1) {
			break;
		}
		/*HANDLE_ERROR( hipEventRecord( stop_kernel, 0 ) );
		 HANDLE_ERROR( hipEventSynchronize( stop_kernel ) );
		 HANDLE_ERROR( hipEventElapsedTime( &elapsedTime_kernel,start_kernel, stop_kernel ) );*/
		//hashes_per_second=(int)((number_of_blocks*threads_per_block*iterations_per_thread)/elapsedTime_kernel*4*1000);
		free(startString);
		free(positionInUsedAlphabet);
	}

	HANDLE_ERROR(hipStreamSynchronize(stream0));
	HANDLE_ERROR(hipStreamSynchronize(stream1));
	HANDLE_ERROR(hipStreamSynchronize(stream2));
	HANDLE_ERROR(hipStreamSynchronize(stream3));
	HANDLE_ERROR(hipEventRecord(stop_overall, 0));
	HANDLE_ERROR(hipEventSynchronize(stop_overall));
	HANDLE_ERROR(
			hipEventElapsedTime(&elapsedTime_overall, start_overall,
					stop_overall));

	printf("\nTime to generate:  %3.1f ms\n", elapsedTime_overall);
	if (match_found[0] == 1) {
		for (uint16_t i = 0; i <= maxLength; i++) {
			result[i] = 0;
		}
		HANDLE_ERROR(
				hipMemcpy(result,dev_result,maxLength,hipMemcpyDeviceToHost));
		cout << result << endl;
	}

	HANDLE_ERROR(hipEventDestroy(start_kernel));
	HANDLE_ERROR(hipEventDestroy(stop_kernel));
	HANDLE_ERROR(hipEventDestroy(start_overall));
	HANDLE_ERROR(hipEventDestroy(stop_overall));
	hipFree(dev_usedAlphabet);
	hipFree(dev_result);
	hipFree(dev_target);
	hipFree(dev_match_found);

	HANDLE_ERROR(hipStreamDestroy(stream0));
	HANDLE_ERROR(hipStreamDestroy(stream1));
	HANDLE_ERROR(hipStreamDestroy(stream2));
	HANDLE_ERROR(hipStreamDestroy(stream3));

	hipDeviceReset();		//important for profiling
}

int main(int argc, char* argv[]) {
	FILE * fp;
	fp = fopen(argv[1], "r");
	if (fp != NULL) {
		char * line = NULL;
		size_t len = 0;
		uint8_t i;
		char digest[64] = { };
		char result[15] = { };
		uint32_t target[8] = { };

		while ((getline(&line, &len, fp)) != -1) {
			for (i = 0; i < 64; i++)
				digest[i] = line[i];
			for (i = 65; i < strlen(line); i++)
				result[i - 65] = line[i];
			printf("\n========== Expected Result: %s\n", result);
			printf("\n%s\n", digest);
			sha256StringToTarget(digest, target);
			sha256bruteforce(target);
			for (i = 0; i < 64; i++)
				digest[i] = 0;
			for (i = 65; i < strlen(line); i++)
				result[i - 65] = 0;
			for (i = 0; i < 8; i++)
				target[i] = 0;
		}

		if (line)
			free(line);
	} else if (strlen(argv[1]) != 64) {
		// Compute SHA256 digest on cpu
		//CPU Hashing begins here
		char output_cpu[64];
		uint8_t sha256sum_cpu[32];
		sha256comp_cpu(argv[1], strlen(argv[1]), sha256sum_cpu);
		sha256_print(sha256sum_cpu, output_cpu);

//		printf("        Input: %s\n", argv[1]);
//		printf("SHA256-Digest: %s\n", output_cpu);
		printf("%s\n", output_cpu);

	} /*else {
	 /* Unsere targets, wurden bei mir alle gefunden. */
	/*
	 hipEvent_t start, stop;
	 float time;
	 hipEventCreate(&start);
	 hipEventCreate(&stop);
	 hipEventRecord(start, 0);
	 hipDeviceSynchronize();

	 uint32_t targets[38][8] = {
	 {0x6a641cbc,0x8ab0602e,0x96c615fb,0x697c6764,0x0e6a1548,0xc1717d0a,0x81cd3bdb,0x64bef43b},  //Durch den bruteforcer hearuasgefunden: deadd
	 {0xcf1e8c45,0x5d16e39f,0xecfd9c20,0x609c46eb,0x9f7566d7,0xc3f7144b,0xe41eb56a,0xc4287826},  //Durch den bruteforcer hearuasgefunden: zxywz
	 {0x3e23e816,0x0039594a,0x33894f65,0x64e1b134,0x8bbd7a00,0x88d42c4a,0xcb73eeae,0xd59c009d},	//b
	 {0xca978112,0xca1bbdca,0xfac231b3,0x9a23dc4d,0xa786eff8,0x147c4e72,0xb9807785,0xafee48bb},	//a
	 {0x34367776,0x2813eaeb,0x65704cc8,0xd9e96f7a,0x444ba0cc,0xa92ff861,0xaf7f6864,0x8b3e6ef1},	//ch
	 {0x1f3ce404,0x15a2081f,0xa3eee75f,0xc39fff8e,0x56c22270,0xd1a978a7,0x249b592d,0xcebd20b4},	//aaaaaaaa
	 {0x38782210,0x12d3785e,0x4f21eef3,0x7119410a,0x7ed8ebb5,0xde28ef82,0xc0cad48d,0x8cdc5d04},	//zzzzzzz	wird nicht gefunden
	 {0x0e10fbe5,0x33c4d8cc,0x73539ce6,0x55057f30,0x92dd975f,0x34aad4f4,0x97f28ad6,0x6ffb503d},	//zzzzzza	wird nicht gefunden
	 {0x7b70d3ab,0x4c764154,0x2e1f158b,0x458eeae7,0xcfb7bdb8,0x15d4110c,0xc6178baf,0xcfdf43f8},	//xxxxxxx
	 {0x67a61945,0x7aae3e86,0x9af3e7c9,0x2078424a,0x773397c1,0x520a9cec,0x76fde54e,0xe8350137},	//qqqqqqq	wird nicht gefunden
	 {0x6ce53691,0xb126808d,0x3745d72b,0x9016384e,0xf0a17400,0x4dc9d3ab,0x151e3c82,0xda186ba5},	//ppppppp	wird nicht gefunden	i>140
	 {0xcc3da533,0x1df88e23,0x2ac0885e,0x142ef9c0,0x27c81432,0x3df9da1c,0xda745b53,0x8f4e950e},	//ooooooo	wird gefunden	i>150
	 {0xbe1f4743,0xf2148891,0x7c4fccfe,0xc99385ee,0xdf039b57,0xb77deba7,0x8b80d5ea,0x04bccb1e},	//nnnnnnn	wird gefunden	i=76
	 {0x800955a7,0xf19d86bd,0x5cca3153,0x050c7299,0xb28f30e9,0x1c7fe854,0xf08a878c,0x158f5aa6},	//lllllll	wird gefunden	i=60
	 {0x5de475c5,0x4f292d35,0x7b4665c4,0xa0667335,0x4d0af583,0xabec2ac5,0x1b752fdf,0x06fcdbbd},	//kkkkkkk	wird gefunden
	 {0x73f5c123,0x3741e5cb,0x12b22c75,0x29f55e46,0xc65b96d7,0xeecce91b,0xc412ca87,0x2c317e45},	//jjjjjjj	wird gefunden
	 {0x15c45977,0xedc54496,0x0301cea5,0xeeb3f6d0,0xac2b96b2,0xd542fd42,0x518ab24d,0xb9c7f829},	//iiiiiii	wird gefunden
	 {0x589f6fec,0xa8b16ba6,0x37fc8a8e,0xa35eea5c,0x224b27e0,0xa65f306e,0x19de14cb,0x0398965a},	//hhhhhhh	wird gefunden
	 {0x85e45110,0x6fb40954,0x4d0edca7,0x0f030bbe,0x905ed7b7,0xd1de93ca,0x20e5390a,0x0a7f3fd5},	//ggggggg	wird gefunden
	 {0x48c1caf3,0x30ea2b39,0xdf2bd04a,0x9b19f344,0x4fd89d61,0xbda17b51,0x1178fd20,0x74b25f9b},	//deadbee	wird gefunden
	 {0x51f36bf2,0x0bc6debb,0x25fe98c5,0x29ecc718,0xc75c2052,0x808521ac,0x4b333a07,0x24881ec5},	//gfedcbb	wird gefunden
	 {0x3de47205,0xe772b39d,0x369b811a,0x8cc515a3,0x1cf31051,0x1bf1f452,0x9b723498,0x8509da55},	//eeeeeee	wird gefunden
	 {0xba24a289,0x0228ef70,0xa16259bb,0x9b72fb1b,0x42ad5f29,0xd7c60a02,0x98456adb,0x4ee11737},	//ddddddd	wird gefunden
	 {0xcd4f3afa,0x6982937e,0x1fcc283c,0x451f1cd9,0xd368e998,0xb554ff7b,0x06eabe6e,0x723a39c2},	//ccccccc	wird gefunden
	 {0xea415a61,0xbd199150,0x84366a0a,0x2fdaebe0,0x70a9c316,0x8877ecdb,0x5e36f490,0x5b5f8aa3},	//bbbbbbb	wird gefunden
	 {0x7d1a5412,0x7b222502,0xf5b79b5f,0xb0803061,0x152a44f9,0x2b37e23c,0x6527baf6,0x65d4da9a},	//abcdefg	wird gefunden
	 {0xcaac75ef,0x1fa69625,0x5f61addf,0x40d7d11d,0x246ed5fe,0xdff2636f,0xbee45e0f,0xe56a1340},	//aaaaaab
	 {0xe4624071,0x4b5db3a2,0x3eee6047,0x9a623efb,0xa4d633d2,0x7fe4f03c,0x904b9e21,0x9a7fbe60},	//aaaaaaa	wird nicht gefunden
	 {0x95fbeb8f,0x769d2c00,0x79d1d113,0x48877da9,0x44aaefab,0xa6ecf9f7,0xf7dab634,0x4ece8605},	//zzzzzz
	 {0xb7fb2176,0x94ae2d30,0x5e766608,0xd250f797,0xdaa984e4,0xac4b5fa6,0x38a729be,0x352f2fcd},	//xxxxxx
	 {0xe2dbf8f5,0xc4cc1514,0x80213d21,0xf95c72aa,0x73a001bc,0xe4915b17,0x691ae409,0x52dcd793},	//ffffff
	 {0x68a55e5b,0x1e43c67f,0x4ef34065,0xa86c4c58,0x3f532ae8,0xe3cda7e3,0x6cc79b61,0x1802ac07},	//zzzzz
	 {0xeaf16bc0,0x7968e013,0xf3f94ab1,0x34247243,0x4a39fc34,0x75f11cf3,0x41a6c396,0x5974f8e9},	//xxxxx
	 {0x99834619,0xb3c16024,0x8b69c7f4,0x2ba868f9,0x45a0ea04,0xcd31cf2f,0x60dc4bc8,0xf7d13b8a},	//fffff
	 {0x2164b17a,0x27fbc64a,0x987a8c8f,0x23341dfd,0x9cef6b90,0x7d75d87b,0x124df41c,0x25812439},	//dzvkb
	 {0x85d0bd50,0xac261c8d,0x3836a816,0x1091b88a,0x12926926,0xcf757e2e,0x2d99c5bd,0x2f452d75},	//deadb
	 {0xba7816bf,0x8f01cfea,0x414140de,0x5dae2223,0xb00361a3,0x96177a9c,0xb410ff61,0xf20015ad}, //abc
	 {0xb8d19b62,0x06b72bf4,0x03a0a87a,0xb21a135a,0x615cc8c5,0x35633000,0x224c831d,0x8a0eb0e3} //dea
	 }; //38 targets

	 uint8_t i,j;
	 uint32_t target[8] = {};
	 for(i = 0; i < 38; i++) {
	 for(j = 0; j < 8; j++) {
	 target[j] = targets[i][j];
	 printf("%08x",target[j]);
	 }
	 printf("\n");
	 sha256bruteforce(target, arguments.verbose);
	 printf("\n\n");
	 }

	 hipEventRecord(stop, 0);
	 hipEventSynchronize(stop);
	 hipEventElapsedTime(&time, start, stop);

	 hipEventDestroy(start);
	 hipEventDestroy(stop);

	 printf("[CPU] took %fms to calcualte.\n", time);

	 /*
	 int i = 2000;
	 while(i-- > 0) {
	 srand(time(NULL ));
	 int someRandValue = rand() % 5;

	 //CPU Hashing begins here
	 char output_cpu[64];
	 uint8_t sha256sum_cpu[32];
	 char* input_string = randstring(7 + someRandValue);
	 sha256comp_cpu(input_string, strlen(input_string), sha256sum_cpu);
	 sha256_print(sha256sum_cpu, output_cpu);

	 printf("\n        Input: %s\n", input_string);
	 printf("SHA256-Digest: %s\n", output_cpu);

	 uint32_t target[8] = { };
	 sha256StringToTarget(output_cpu, target);
	 sha256bruteforce(target, arguments.verbose);
	 }
	 } */
	else {
		uint32_t target[8] = { };
		sha256StringToTarget(argv[1], target);
		sha256bruteforce(target);
	}

	return EXIT_SUCCESS;
}
